
#include <hip/hip_runtime.h>
#include <iostream>

#include <cstdint>      // Data types
#include <iostream>     // File operations

// #define M 512       // Lenna width
// #define N 512       // Lenna height
#define M 941       // VR width
#define N 704       // VR height
#define C 3         // Colors
#define OFFSET 15   // Header length

__global__ void invert(uint8_t* data, int blocks) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int max = M*N*C;
    int grid_size = blockDim.x * blocks;
    int k = 0;
    while(index + k * grid_size < max){
        int pixel = index + k * grid_size;
        if(pixel % 3 == 0 && data[pixel] >= 100 && data[pixel] <= 200) data[pixel] = (data[pixel]%25) * 10;
        else data[pixel] = 255 - data[pixel];
        k++;
    }
}

// https://imagetostl.com/view-ppm-online

uint8_t* get_image_array(void){
    /*
     * Get the data of an (RGB) image as a 1D array.
     *
     * Returns: Flattened image array.
     *
     * Noets:
     *  - Images data is flattened per color, column, row.
     *  - The first 3 data elements are the RGB components
     *  - The first 3*M data elements represent the firts row of the image
     *  - For example, r_{0,0}, g_{0,0}, b_{0,0}, ..., b_{0,M}, r_{1,0}, ..., b_{b,M}, ..., b_{N,M}
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile=fopen("./input_image.ppm","rb");
    if(imageFile==NULL){
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }

    // Initialize empty image array
    uint8_t* image_array = (uint8_t*)malloc(M*N*C*sizeof(uint8_t)+OFFSET);

    // Read the image
    fread(image_array, sizeof(uint8_t), M*N*C*sizeof(uint8_t)+OFFSET, imageFile);

    // Close the file
    fclose(imageFile);

    // Move the starting pointer and return the flattened image array
    return image_array + OFFSET;
}


void save_image_array(uint8_t* image_array){
    /*
     * Save the data of an (RGB) image as a pixel map.
     *
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     *
     */
    // Try opening the file
    FILE *imageFile;
    imageFile=fopen("./output_image.ppm","wb");
    if(imageFile==NULL){
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }


    // Configure the file
    fprintf(imageFile,"P6\n");               // P6 filetype
    fprintf(imageFile,"%d %d\n", M, N);      // dimensions
    fprintf(imageFile,"255\n");              // Max pixel

    // Write the image
    fwrite(image_array, 1, M*N*C, imageFile);

    // Close the file
    fclose(imageFile);
}

void process(uint8_t* image_array, int blocks, int threads) {


    // Allocate output
//    uint8_t* new_image_array = (uint8_t*)malloc(M*N*C);

    // Convert to grayscale using only the red color component
//    for(int i=0; i<M*N*C; i++){
//        new_image_array[i] = image_array[i/3*3];
//    }

    uint8_t* data;
    hipMalloc(&data, M*N*C*sizeof(uint8_t));
    hipMemcpy( data, image_array, M*N*C*sizeof(uint8_t), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start);

    invert<<<blocks,threads>>>(data, blocks);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds;

//    cudaMemcpy( new_image_array, data, M*N*C*sizeof(uint8_t), cudaMemcpyDeviceToHost);

    // Save the image
//    save_image_array(new_image_array);

//    free(image_array);
//    free(new_image_array);
    hipFree(data);
}

int main (void) {
    // Read the image
    uint8_t* image_array = get_image_array();
    process(image_array, 1, 192);
    std::cout << std::endl;
    std::cout << "threads per block;1;2;4;8;16;32" << std::endl;
    for(int threads = 16; threads <= 1024; threads++) {
        std::cout << threads;
        for(int blocks = 1; blocks <= 32; blocks*=2) {
            std::cout << ";";
            process(image_array, blocks, threads);
        }
        std::cout << std::endl;
    }
    //free(image_array);
    return 0;
}